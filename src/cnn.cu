#include "hip/hip_runtime.h"
#include<cnn.h>

#define CUDA_CHECK(call)                                                      \
    do {                                                                      \
        hipError_t err = call;                                               \
        if (err != hipSuccess) {                                             \
            std::cerr << "CUDA error in " << __FILE__ << ":" << __LINE__     \
                      << " - " << hipGetErrorString(err) << std::endl;      \
            return 0.0;                                                       \
        }                                                                     \
    } while (0)


namespace image_processing {
    
    double ApplyConv2DKernelCuda(const std::vector<unsigned char>& image, 
                                std::vector<unsigned char>& result, 
                                uint32_t width, 
                                uint32_t height,
                                uint32_t poolWidth,
                                uint32_t poolHeight,
                                const std::vector<float>& kernelX, 
                                const std::vector<float>& kernelY, 
                                int kernel_size) {
        unsigned char* d_image = nullptr;
        unsigned char* d_result = nullptr;
        float* d_tmp = nullptr;
        float* d_kernelX = nullptr;
        float* d_kernelY = nullptr;

        size_t image_size = width * height * sizeof(unsigned char);
        size_t tmp_size = width * height * sizeof(float);
        size_t result_size = static_cast<uint32_t>((width / poolWidth) * (height / poolHeight)) * sizeof(unsigned char);
        size_t kernel_size_bytes = kernel_size * kernel_size * sizeof(float);

        CUDA_CHECK(hipMalloc(&d_tmp, tmp_size));
        CUDA_CHECK(hipMalloc(&d_image, image_size));
        CUDA_CHECK(hipMalloc(&d_result, result_size));
        CUDA_CHECK(hipMalloc(&d_kernelX, kernel_size_bytes));
        CUDA_CHECK(hipMalloc(&d_kernelY, kernel_size_bytes));

        CUDA_CHECK(hipMemcpy(d_image, image.data(), image_size, hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(d_kernelX, kernelX.data(), kernel_size_bytes, hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(d_kernelY, kernelY.data(), kernel_size_bytes, hipMemcpyHostToDevice));

        dim3 block_dim(kBlockSize, kBlockSize);
        dim3 grid_dim((width + kBlockSize - 1) / kBlockSize, (height + kBlockSize - 1) / kBlockSize);

        bool useSharedMem = true;  // or false

        size_t sharedMemSize = 0;
        if (useSharedMem) {
            sharedMemSize = (kBlockSize + kernel_size - 1) *
                            (kBlockSize + kernel_size - 1) * sizeof(unsigned char);
        }

        std::cout << "Launching kenrnel with block size: " << kBlockSize << " and " << "grid size: " << (width + kBlockSize - 1) / kBlockSize << std::endl;

        auto start = std::chrono::high_resolution_clock::now();
        Conv2DKernel<<<grid_dim, block_dim, sharedMemSize>>>(d_image, d_tmp, width, height,
                                                             d_kernelX, d_kernelY, kernel_size,
                                                             useSharedMem);
        ReLUKernel<<<grid_dim, block_dim>>>(d_tmp, width, height);
        MaxPool2D<<<grid_dim, block_dim>>>(d_tmp, d_result, width, height, poolWidth, poolHeight);
        auto end = std::chrono::high_resolution_clock::now();

        CUDA_CHECK(hipGetLastError());
        CUDA_CHECK(hipDeviceSynchronize());

        CUDA_CHECK(hipMemcpy(result.data(), d_result, result_size, hipMemcpyDeviceToHost));

        hipFree(d_image);
        hipFree(d_tmp);
        hipFree(d_result);
        hipFree(d_kernelX);
        hipFree(d_kernelY);

        return std::chrono::duration<double>(end - start).count();
    }

    __global__ void Conv2DKernel(const unsigned char* image, 
                                 float* result, 
                                 uint32_t width, 
                                 uint32_t height, 
                                 const float* kernelX, 
                                 const float* kernelY, 
                                 int kernel_size,
                                 bool useSharedMem) {
        int tx = threadIdx.x;
        int ty = threadIdx.y;

        int x = blockIdx.x * blockDim.x + tx;
        int y = blockIdx.y * blockDim.y + ty;

        int half_kernel = kernel_size / 2;

        if (useSharedMem) {
            extern __shared__ unsigned char sharedImage[];

            int shared_width = blockDim.x + kernel_size - 1;
            int shared_height = blockDim.y + kernel_size - 1;

            // Load the shared memory tile
            for (int dy = ty; dy < shared_height; dy += blockDim.y) {
                for (int dx = tx; dx < shared_width; dx += blockDim.x) {
                    int global_x = min(max(blockIdx.x * blockDim.x + dx - half_kernel, 0), width - 1);
                    int global_y = min(max(blockIdx.y * blockDim.y + dy - half_kernel, 0), height - 1);
                    sharedImage[dy * shared_width + dx] = image[global_y * width + global_x];
                }
            }

            __syncthreads();

            if (x < width && y < height) {
                float gradientX = 0.0f, gradientY = 0.0f;

                for (int ky = 0; ky < kernel_size; ky++) {
                    for (int kx = 0; kx < kernel_size; kx++) {
                        int sx = tx + kx;
                        int sy = ty + ky;
                        float pixel = static_cast<float>(sharedImage[sy * shared_width + sx]);

                        gradientX += pixel * kernelX[ky * kernel_size + kx];
                        gradientY += pixel * kernelY[ky * kernel_size + kx];
                    }
                }

                result[y * width + x] = sqrtf(gradientX * gradientX + gradientY * gradientY);
            }
        } else {
            // Fallback to global memory
            if (x < width && y < height) {
                float gradientX = 0.0f, gradientY = 0.0f;

                for (int ky = -half_kernel; ky <= half_kernel; ky++) {
                    for (int kx = -half_kernel; kx <= half_kernel; kx++) {
                        int ix = min(max(x + kx, 0), width - 1);
                        int iy = min(max(y + ky, 0), height - 1);
                        float pixel = static_cast<float>(image[iy * width + ix]);

                        gradientX += pixel * kernelX[(ky + half_kernel) * kernel_size + (kx + half_kernel)];
                        gradientY += pixel * kernelY[(ky + half_kernel) * kernel_size + (kx + half_kernel)];
                    }
                }

                result[y * width + x] = sqrtf(gradientX * gradientX + gradientY * gradientY);
            }
        }
    }
    
    __global__ void ReLUKernel(float* image, int width, int height) {
        int x = blockIdx.x * blockDim.x + threadIdx.x;
        int y = blockIdx.y * blockDim.y + threadIdx.y;

        if (x < width && y < height) {
            int idx = y * width + x;
            image[idx] = min(max(static_cast<int>(roundf(image[idx])), 0), 255);
        }
    }

    __global__ void MaxPool2D(float* input, unsigned char* result, int width, int height, int poolWidth, int poolHeight) {
        int x = blockIdx.x * blockDim.x + threadIdx.x;
        int y = blockIdx.y * blockDim.y + threadIdx.y;

        if (x * poolWidth >= width || y * poolHeight >= height) return;

        float maxVal = -FLT_MAX;
        for (int i = 0; i < poolHeight; ++i) {
            for (int j = 0; j < poolWidth; ++j) {
                int curX = x * poolWidth + j;
                int curY = y * poolHeight + i;
                if (curX < width && curY < height) {
                    float val = input[curY * width + curX];
                    if (val > maxVal) {
                        maxVal = val;
                    }
                }
            }
        }
        result[y * (width / poolWidth) + x] = static_cast<unsigned char>(maxVal);
    }

    void GenerateSobelKernels(int size, std::vector<float>& kernelX, std::vector<float>& kernelY) {
        // Ensure the size is odd (e.g., 3x3, 5x5, etc.)
        if (size % 2 == 0) {
            std::cerr << "Kernel size must be odd!" << std::endl;
            return;
        }

        int mid = size / 2;

        // Generate horizontal (X) kernel
        for (int i = 0; i < size; ++i) {
            for (int j = 0; j < size; ++j) {
                kernelX[i * size + j] = (j - mid) * (mid - abs(i - mid));
            }
        }

        // Generate vertical (Y) kernel
        for (int i = 0; i < size; ++i) {
            for (int j = 0; j < size; ++j) {
                kernelY[i * size + j] = (i - mid) * (mid - abs(j - mid));
            }
        }
    }

}